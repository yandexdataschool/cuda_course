
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <time.h>

#define BLOCK_SIZE 256

static constexpr int numIterations = 100;
static constexpr int numValuesToPrint = 10;

__global__ void func1_kernel(const float* in, float* out, int numElements)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < numElements)
    {
        float value = in[i];
        for (int iter = 0; iter < numIterations; iter++)
        {
            value = sinf(value);
        }
        out[i] = value;
    }
}

__global__ void func2_kernel(const float* in1, const float* in2, float* out, int numElements)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < numElements)
    {
        float value1 = in1[numElements - i - 1];
        float value2 = in2[i];
        for (int iter = 0; iter < numIterations; iter++)
        {
            value2 = -sinf(value2);
        }
        out[i] = value1 + value2;
    }
}

int main(int argc, char* argv[])
{

    int numElements = (argc > 1) ? atoi(argv[1]) : 1000000;

    printf("Transforming %d values.\n", numElements);

    float* h_data1   = (float*)calloc(numElements, sizeof(float));
    float* h_data2   = (float*)calloc(numElements, sizeof(float));

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        h_data1[i] = float(rand())/float(RAND_MAX + 1.0);
        h_data2[i] = float(rand())/float(RAND_MAX + 1.0);
    }

    int threadsPerBlock = BLOCK_SIZE;
    int numBlocks = numElements/BLOCK_SIZE + 1;

    float* d_data1;
    float* d_data2;

    hipMalloc((void**)&d_data1, numElements*sizeof(float));
    hipMalloc((void**)&d_data2, numElements*sizeof(float));

    // Timing
    clock_t start = clock();

    hipMemcpy(d_data1, h_data1, numElements*sizeof(float), hipMemcpyHostToDevice);
    func1_kernel<<<numBlocks, threadsPerBlock>>>(d_data1, d_data1, numElements);
    hipMemcpy(h_data1, d_data1, numElements*sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(d_data2, h_data2, numElements*sizeof(float), hipMemcpyHostToDevice);
    func2_kernel<<<numBlocks, threadsPerBlock>>>(d_data1, d_data2, d_data2, numElements);
    hipMemcpy(h_data2, d_data2, numElements*sizeof(float), hipMemcpyDeviceToHost);

    // Timing
    clock_t finish = clock();

    printf("The results are:\n");
    for (int i = 0; i < numValuesToPrint; i++)
    {
        printf("%f, %f\n", h_data1[i], h_data2[i]);
    }
    printf("...\n");
    for (int i = numElements - numValuesToPrint; i < numElements; i++)
    {
        printf("%f, %f\n", h_data1[i], h_data2[i]);
    }
    double sum1 = 0.0;
    double sum2 = 0.0;
    for (int i = 0; i < numElements; i++)
    {
        sum1 += h_data1[i];
        sum2 += h_data2[i];
    }
    printf("The summs are: %f and %f\n", sum1, sum2);

    printf("It took %f seconds\n", (double)(finish - start) / CLOCKS_PER_SEC);

    // Release the memory
    free(h_data1);
    free(h_data2);
    
    hipFree(d_data1);
    hipFree(d_data2);

    return 0;
}